
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <math.h>
#include <hipblas.h>


int main() {
    int Width = 100;
    float *M, *N, *P;
    hipMallocManaged(&M, sizeof(float) * Width*Width);
    hipMallocManaged(&N, sizeof(float) * Width*Width);
    hipMallocManaged(&P, sizeof(float) * Width*Width);

    //initialize inputs
    for (int i = 0; i < Width*Width; i++){
        M[i] = 1;
        N[i] = 1;
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    const float alpha = 1.0f;
    const float beta = 0.0f;
    clock_t a = clock();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Width, Width, Width, &alpha, N, Width, M, Width, &beta, P, Width);
    hipDeviceSynchronize();
    clock_t b = clock() - a;
    for (int i = 0; i < Width*Width; i++){
        printf("%f ", P[i]);
    }
    printf("time: %f ", (float) b/CLOCKS_PER_SEC);
    printf("\n");

    printf("%s\n", hipGetErrorString(hipGetLastError()));
    hipFree(M);
    hipFree(N);
    hipFree(P);
    return 0;
}
